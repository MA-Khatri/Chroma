#include "hip/hip_runtime.h"
#include "utils.cuh"

namespace otx
{
	extern "C" __global__ void __closesthit__radiance()
	{
		const SBTData& sbtData = *(const SBTData*)optixGetSbtDataPointer();
		PRD_Radiance& prd = *getPRD<PRD_Radiance>();

		const int primID = optixGetPrimitiveIndex();
		const int3 index = sbtData.index[primID];
		float2 uv = optixGetTriangleBarycentrics();

		/* Use the stored color in the sbtData as the brightness of the light */
		float3 lightColor = sbtData.reflectionColor;

		/* If this light has a diffuse texture, scale the lightColor by that texture (creating a textured light) */
		float2 tc = TexCoord(uv, sbtData.texCoord[index.x], sbtData.texCoord[index.y], sbtData.texCoord[index.z]);
		if (sbtData.hasDiffuseTexture)
		{
			float4 tex = tex2D<float4>(sbtData.diffuseTexture, tc.x, tc.y);
			lightColor *= make_float3(tex.x, tex.y, tex.z);
		}
		prd.radiance = lightColor;

		/* Terminate ray */
		prd.done = true;

		/* If this is the first intersection of the ray, set the albedo and normal */
		if (prd.depth == 0)
		{
			prd.albedo = sbtData.reflectionColor;

			/* Use shading normal if available, else use geometry normal */
			const float3& v0 = sbtData.position[index.x];
			const float3& v1 = sbtData.position[index.y];
			const float3& v2 = sbtData.position[index.z];
			float3 N = (sbtData.normal)
				? InterpolateNormals(uv, sbtData.normal[index.x], sbtData.normal[index.y], sbtData.normal[index.z])
				: cross(v1 - v0, v2 - v0);

			/* Compute world-space normal and normalize */
			N = normalize(optixTransformNormalFromObjectToWorldSpace(N));
			prd.normal = N;
		}
	}


	extern "C" __global__ void __anyhit__radiance()
	{
		// TODO?
	}
}