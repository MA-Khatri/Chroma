#include "hip/hip_runtime.h"
#include "utils.cuh"

namespace otx
{
	extern "C" __global__ void __closesthit__shadow()
	{
		/* Not going to be used... */
	}

	extern "C" __global__ void __anyhit__shadow()
	{
		/* Not going to be used... */
	}

	extern "C" __global__ void __miss__shadow()
	{
		/* Nothing was hit so the light is visible */
		PRD_Shadow& prd = *(PRD_Shadow*)getPRD<PRD_Shadow>();
		prd.radiance = make_float3(5.0f);
		prd.reachedLight = true;
	}
}