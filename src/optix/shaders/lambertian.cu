#include "hip/hip_runtime.h"
#include "utils.cuh"

namespace otx
{
	extern "C" __global__ void __closesthit__radiance()
	{
		const SBTData& sbtData = *(const SBTData*)optixGetSbtDataPointer();
		PRD_Radiance& prd = *getPRD<PRD_Radiance>();

		const int primID = optixGetPrimitiveIndex();
		const int3 index = sbtData.index[primID];
		float2 uv = optixGetTriangleBarycentrics();
		float3 outDir = -optixGetWorldRayDirection();

		/* === Compute normal === */
		/* Use shading normal if available, else use geometry normal */
		const float3& v0 = sbtData.position[index.x];
		const float3& v1 = sbtData.position[index.y];
		const float3& v2 = sbtData.position[index.z];
		float3 N = (sbtData.normal) ? InterpolateNormals(uv, sbtData.normal[index.x], sbtData.normal[index.y], sbtData.normal[index.z]) : cross(v1 - v0, v2 - v0);

		/* Compute world-space normal and normalize */
		N = normalize(optixTransformNormalFromObjectToWorldSpace(N));

		/* Face forward normal */
		if (dot(outDir, N) < 0.0f) N = -N;

		/* Update the hit position */
		prd.origin = FrontHitPosition(N);

		/* Update the basis for this intersection */
		prd.basis = OrthonormalBasis(N);

		/* Generate a new sample direction (in_direction) */
		prd.out_direction = prd.in_direction;
		prd.in_direction = prd.basis.Local(prd.random.RandomOnUnitCosineHemisphere());

		/* Default diffuse color if no diffuse texture */
		float3 diffuseColor = sbtData.reflectionColor;

		/* === Sample diffuse texture === */
		float2 tc = TexCoord(uv, sbtData.texCoord[index.x], sbtData.texCoord[index.y], sbtData.texCoord[index.z]);
		if (sbtData.hasDiffuseTexture)
		{
			float4 tex = tex2D<float4>(sbtData.diffuseTexture, tc.x, tc.y);
			diffuseColor = make_float3(tex.x, tex.y, tex.z);
		}

		/* If this is the first intersection of the ray, set the albedo and normal */
		if (prd.depth == 0)
		{
			prd.albedo = diffuseColor;
			prd.normal = N;
		}

		/* Update throughput */
		float bsdf = M_1_PIf;
		float pdf = CosineHemispherePDF(prd.in_direction, N);
		prd.throughput *= diffuseColor * bsdf * max(dot(prd.in_direction, N), 0.0f) / pdf;
	}


	extern "C" __global__ void __anyhit__radiance()
	{
		// TODO?
	}
}