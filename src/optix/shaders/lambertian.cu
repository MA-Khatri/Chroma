#include "hip/hip_runtime.h"
#include "utils.cuh"

namespace otx
{
	__forceinline__ __device__ float3 Sample(PRD_Radiance& prd)
	{
		return prd.basis.Local(prd.random.RandomOnUnitCosineHemisphere());
	}

	__forceinline__ __device__ float3 Eval(PRD_Radiance& prd, float3 indir, float3 outdir)
	{
		const SBTData& sbtData = *prd.sbtData;
		const int3 index = sbtData.index[prd.primID];

		/* Default diffuse color if no diffuse texture */
		float3 diffuseColor = sbtData.reflectionColor;

		/* === Sample diffuse texture === */
		float2 tc = TexCoord(prd.uv, sbtData.texCoord[index.x], sbtData.texCoord[index.y], sbtData.texCoord[index.z]);
		if (sbtData.hasDiffuseTexture)
		{
			float4 tex = tex2D<float4>(sbtData.diffuseTexture, tc.x, tc.y);
			diffuseColor = make_float3(tex.x, tex.y, tex.z);
		}

		/* If this is the first intersection of the ray, set the albedo and normal */
		if (prd.depth == 0)
		{
			prd.albedo = diffuseColor;
			prd.normal = prd.basis.w;
		}

		return diffuseColor * max(dot(indir, prd.basis.w), 0.0f) * M_1_PIf;
	}


	__forceinline__ __device__ float PDF(PRD_Radiance& prd, float3 w)
	{
		/* I.e., cosine hemisphere pdf */
		return max(dot(w, prd.basis.w), 0.0f) * M_1_PIf;
	}


	extern "C" __global__ void __closesthit__radiance()
	{
		PRD_Radiance& prd = *getPRD<PRD_Radiance>();
		prd.sbtData = (const SBTData*)optixGetSbtDataPointer();
		const SBTData& sbtData = *prd.sbtData;
		prd.Sample = CALLABLE_LAMBERTIAN_SAMPLE;
		prd.Eval = CALLABLE_LAMBERTIAN_EVAL;
		prd.PDF = CALLABLE_LAMBERTIAN_PDF;
		
		prd.primID = optixGetPrimitiveIndex();
		const int3 index = sbtData.index[prd.primID];
		prd.uv = optixGetTriangleBarycentrics();
		float3 outDir = -optixGetWorldRayDirection();

		/* === Compute normal === */
		/* Use shading normal if available, else use geometry normal */
		const float3& v0 = sbtData.position[index.x];
		const float3& v1 = sbtData.position[index.y];
		const float3& v2 = sbtData.position[index.z];
		float3 N = (sbtData.normal) 
			? InterpolateNormals(prd.uv, sbtData.normal[index.x], sbtData.normal[index.y], sbtData.normal[index.z]) 
			: cross(v1 - v0, v2 - v0);

		/* Compute world-space normal and normalize */
		N = normalize(optixTransformNormalFromObjectToWorldSpace(N));

		/* Face forward normal */
		if (dot(outDir, N) < 0.0f) N = -N;

		/* Update the hit position */
		prd.origin = FrontHitPosition(N);

		/* Update the basis for this intersection */
		prd.basis = OrthonormalBasis(N);

		/* Generate a new sample direction (in_direction) */
		prd.out_direction = prd.in_direction;
		prd.in_direction = Sample(prd);
		prd.specular = false;

		/* Update throughput */
		float3 bsdf = Eval(prd, prd.in_direction, prd.out_direction);
		float pdf = PDF(prd, prd.in_direction);
		prd.throughput *= bsdf / pdf;
		prd.pdf *= pdf;

		/* Store the world space positions of the hit triangle vertices */
		prd.p0 = optixTransformPointFromObjectToWorldSpace(v0);
		prd.p1 = optixTransformPointFromObjectToWorldSpace(v1);
		prd.p2 = optixTransformPointFromObjectToWorldSpace(v2);
	}


	extern "C" __global__ void __anyhit__radiance()
	{
		// TODO?
	}


	extern "C" __device__ float3 __direct_callable__sample(PRD_Radiance & prd)
	{
		return Sample(prd);
	}


	extern "C" __device__ float3 __direct_callable__eval(PRD_Radiance& prd, float3 indir, float3 outdir)
	{
		return Eval(prd, indir, outdir);
	}


	extern "C" __device__ float __direct_callable__pdf(PRD_Radiance& prd, float3 w)
	{
		return PDF(prd, w);
	}
}