#include "hip/hip_runtime.h"
#include "utils.cuh"

namespace otx
{
	/* ================================ */
	/* === Ray Generation Functions === */
	/* ================================ */

	__forceinline__ __device__ float2 GenerateScreenPosition(int ix, int iy, Random& random)
	{
		/* Normalized screen plane position in [0, 1]^2 with randomized sub-pixel position */
		return (make_float2((float)ix, (float)iy) + random.RandomSample2D()) / make_float2(optixLaunchParams.frame.size.x, optixLaunchParams.frame.size.y);
	}


	__forceinline__ __device__ void GenerateCameraRay(PRD_Radiance& prd, float2 screen)
	{
		/* Get the camera from launchParams */
		const auto& camera = optixLaunchParams.camera;

		switch (optixLaunchParams.camera.projectionMode)
		{
		case PROJECTION_MODE_PERSPECTIVE:
		{
			prd.origin = camera.position;
			prd.in_direction = normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical);
			break;
		}
		case PROJECTION_MODE_ORTHOGRAPHIC:
		{
			prd.origin = camera.position + (screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical;
			prd.in_direction = camera.direction;
			break;
		}
		case PROJECTION_MODE_THIN_LENS:
		{
			float2 p = prd.random.RandomInUnitDisk();
			float3 orgOffset = (p.x * camera.defocusDiskU) + (p.y * camera.defocusDiskV);
			prd.origin = camera.position + orgOffset;
			prd.in_direction = normalize(camera.direction + ((screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical) - orgOffset);
			break;
		}
		}
	}


	/* =================== */
	/* === Integrators === */
	/* =================== */

	/* === Path Integrator and related helpers === */
	/*
	 * The version of the path integrator that was (finally) working correctly was based on this stack exchange post:
	 * https://computergraphics.stackexchange.com/questions/5152/progressive-path-tracing-with-explicit-light-sampling
	 */

	__forceinline__ __device__ float3 CalculateDirectLightSamplePDF(PRD_Radiance& prd, PRD_Shadow& shadowRay, int l, float3& lightSampleDirection, float& distance)
	{
		/* Get the corresponding light */
		MISLight light;
		if (l == 0) light.type = LIGHT_TYPE_BACKGROUND;
		else light = *(optixLaunchParams.lights + (l - 1) * sizeof(MISLight));

		float3 lightRadiance = make_float3(0.0f);

		switch (light.type)
		{
		case LIGHT_TYPE_BACKGROUND:
		{
			/* Background sample can be anywhere on the unit sphere */
			lightSampleDirection = prd.random.RandomOnUnitSphere();
			distance = 1e20f;

			/* Probability of sampling this direction of the background */
			float cosTheta = max(dot(lightSampleDirection, make_float3(0.0f, 0.0f, 1.0f)), 0.0f);
			shadowRay.pdf = cosTheta > 0.0f ? 1.0f / cosTheta : 0.0f;

			lightRadiance = optixDirectCall<float3, float3>(CALLABLE_SAMPLE_BACKGROUND, lightSampleDirection);

			break;
		}
		case LIGHT_TYPE_AREA:
		{
			/* Sample a point on the triangle */
			float3 a = light.p1 - light.p0;
			float3 b = light.p2 - light.p0;

			float2 uv = prd.random.RandomSample2D();
			if (uv.x + uv.y > 1.0f) uv = make_float2(1.0f - uv.x, 1.0f - uv.y);

			float3 lightSamplePosition = light.p0 + a * uv.x + b * uv.y;
			float3 lsd = lightSamplePosition - prd.origin;
			float distance2 = dot(lsd, lsd);
			distance = sqrtf(distance2);
			lightSampleDirection = lsd / distance;

			/* Probability of sampling this point on the triangle */
			float3 lightNormal = InterpolateNormals(uv, light.n0, light.n1, light.n2);
			float cosTheta = max(dot(lightSampleDirection, -lightNormal), 0.0f);
			shadowRay.pdf = cosTheta > 0.0f ? distance2 / (light.area * cosTheta) : 0.0f;

			lightRadiance = light.emissionColor;

			break;
		}
		case LIGHT_TYPE_DELTA:
		{
			//TODO
			break;
		}
		}

		/* Subtract a small eps to prevent counting the intersection with the light surface itself */
		distance -= RAY_EPS;

		return lightRadiance;
	}


	__forceinline__ __device__ float3 CalculateBSDFLightSamplePDF(PRD_Radiance& prd, PRD_Shadow& shadowRay, int l, float3 lightSampleDirection, float& distance)
	{
		/* Note: The PDFs here will be the inverse of the corresponding PDFs in DirectLightSamplePDF */

		/* Get the corresponding light */
		MISLight light;
		if (l == 0) light.type = LIGHT_TYPE_BACKGROUND;
		else light = *(optixLaunchParams.lights + (l - 1) * sizeof(MISLight));

		float3 lightRadiance = make_float3(0.0f);

		switch (light.type)
		{
		case LIGHT_TYPE_BACKGROUND:
		{
			distance = 1e20f;

			/* Probability of sampling this direction of the background */
			float cosTheta = max(dot(lightSampleDirection, make_float3(0.0f, 0.0f, 1.0f)), 0.0f);
			shadowRay.pdf = cosTheta;

			lightRadiance = optixDirectCall<float3, float3>(CALLABLE_SAMPLE_BACKGROUND, lightSampleDirection);

			break;
		}
		case LIGHT_TYPE_AREA:
		{
			/* Trace the ray and see if it will intersect this triangle */
			/* https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm */
			float3 e1 = light.p1 - light.p0;
			float3 e2 = light.p2 - light.p0;
			float3 rce2 = cross(lightSampleDirection, e2);
			float det = dot(e1, rce2);

			if (det > -RAY_EPS && det < RAY_EPS)
			{ /* Ray is parallel */
				shadowRay.pdf = 0.0f;
				return lightRadiance;
			}

			float inv_det = 1.0f / det;
			float3 s = prd.origin - light.p0;
			float u = inv_det * dot(s, rce2);
			if (u < 0.0f || u > 1.0f)
			{ /* Out of bounds of triangle */
				shadowRay.pdf = 0.0f;
				return lightRadiance;
			}

			float3 sce1 = cross(s, e1);
			float v = inv_det * dot(lightSampleDirection, sce1);
			if (v < 0.0f || u + v > 1.0f)
			{ /* Out of bounds of triangle */
				shadowRay.pdf = 0.0f;
				return lightRadiance;
			}

			distance = inv_det * dot(e2, sce1);
			
			if (distance < RAY_EPS)
			{
				shadowRay.pdf = 0.0f;
				return lightRadiance;
			}

			/* Probability of sampling this point on the triangle */
			float3 lightNormal = InterpolateNormals(make_float2(u, v), light.n0, light.n1, light.n2);
			float cosTheta = max(dot(lightSampleDirection, -lightNormal), 0.0f);
			shadowRay.pdf = (light.area * cosTheta) / (distance * distance);

			lightRadiance = light.emissionColor;

			break;
		}
		case LIGHT_TYPE_DELTA:
		{
			//TODO
			break;
		}
		}

		/* Subtract a small eps to prevent counting the intersection with the light surface itself */
		distance -= RAY_EPS;

		return lightRadiance;
	}


	__forceinline__ __device__ float3 ImportanceSampleLight(PRD_Radiance& prd)
	{
		/* Initializing... */
		float3 directLighting = make_float3(0.0f);

		float3 bsdf;
		float scatteringPDF;

		float3 lightSampleDirection;
		float distance;

		/* Initialize a shadow ray... */
		PRD_Shadow shadowRay;
		shadowRay.throughput = make_float3(0.0f);
		shadowRay.pdf = 0.0f;
		shadowRay.reached_light = false;


		/* We add 1 light for the back ground... */
		int nLights = optixLaunchParams.nLights + 1;

		/* Choose a light to sample -- we can later use more advanced methods such as choosing based on light power */
		int l = (int)((float)nLights * prd.random());

		/* To speed up the frame rate, we randomly choose whether to sample the light directly or the via the bsdf */
		float lsr = optixLaunchParams.lightSampleRate;

		/* ============================= */
		/* === DIRECT LIGHT SAMPLING === */
		/* ============================= */

		/* Only sample the light directly if this was not a specular bounce */
		if (!prd.specular && prd.random() < lsr)
		{
			/*
			 * Calculate the PDF of sampling the chosen light (stored in shadowRay.pdf),
			 * the radiance of the light, and the distance to the chosen sample point
			 */
			float3 lightRadiance = CalculateDirectLightSamplePDF(prd, shadowRay, l, lightSampleDirection, distance);

			if (shadowRay.pdf > 0.0f)
			{
				/* Evaluate the BSDF for the chosen light sample direction */
				bsdf = optixDirectCall<float3, PRD_Radiance&, float3, float3>(prd.Eval, prd, lightSampleDirection, prd.out_direction);

				/* Probability of light scattering in the chosen light sample direction */
				scatteringPDF = optixDirectCall<float, PRD_Radiance&, float3>(prd.PDF, prd, lightSampleDirection);

				if (scatteringPDF > 0.0f && (bsdf.x > 0.0f || bsdf.y > 0.0f || bsdf.z > 0.0f))
				{
					/* Launch the shadow ray towards the selected light */
					uint32_t s0, s1;
					packPointer(&shadowRay, s0, s1);
					optixTrace(
						optixLaunchParams.traversable,
						prd.origin, /* I.e., last hit position of the primary ray path */
						lightSampleDirection,
						0.0f, /* prd.origin should already be offset */
						distance,
						0.0f, /* ray time */
						OptixVisibilityMask(255),
						OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
						RAY_TYPE_SHADOW,
						RAY_TYPE_COUNT,
						RAY_TYPE_SHADOW,
						s0, s1
					);

					if (shadowRay.reached_light)
					{
						directLighting += powerHeuristic(shadowRay.pdf, scatteringPDF) * lightRadiance * bsdf / shadowRay.pdf;
					}
				}
			}
		}
		else
		{
			/* =========================== */
			/* === BSDF LIGHT SAMPLING === */
			/* =========================== */

			/* Generate a sample direction from the bsdf */
			lightSampleDirection = optixDirectCall<float3, PRD_Radiance&>(prd.Sample, prd);

			/* Evaluate the bsdf for the chosen direction */
			bsdf = optixDirectCall<float3, PRD_Radiance&, float3, float3>(prd.Eval, prd, lightSampleDirection, prd.out_direction);

			/* Evaluate the pdf for the chosen direction */
			scatteringPDF = optixDirectCall<float, PRD_Radiance&, float3>(prd.PDF, prd, lightSampleDirection);
		
			if (scatteringPDF > 0.0f && (bsdf.x > 0.0f || bsdf.y > 0.0f || bsdf.z > 0.0f))
			{
				float3 lightRadiance = CalculateBSDFLightSamplePDF(prd, shadowRay, l, lightSampleDirection, distance);

				/* No bsdf sample if pdf is leq 0 */
				if (shadowRay.pdf <= 0.0f) return (float)nLights * directLighting;

				/* Launch the shadow ray towards the selected light */
				uint32_t s0, s1;
				packPointer(&shadowRay, s0, s1);
				optixTrace(
					optixLaunchParams.traversable,
					prd.origin, /* I.e., last hit position of the primary ray path */
					lightSampleDirection,
					0.0f, /* prd.origin should already be offset */
					distance,
					0.0f, /* ray time */
					OptixVisibilityMask(255),
					OPTIX_RAY_FLAG_DISABLE_ANYHIT | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
					RAY_TYPE_SHADOW,
					RAY_TYPE_COUNT,
					RAY_TYPE_SHADOW,
					s0, s1
				);

				if (shadowRay.reached_light)
				{
					directLighting += powerHeuristic(scatteringPDF, shadowRay.pdf) * bsdf * lightRadiance / scatteringPDF;
				}
			}
		}

		/* We multiply by nLights to compensate for choosing this light */
		return (float)nLights * directLighting;
	}


	__forceinline__ __device__ void PathIntegrator(PRD_Radiance& prd)
	{
		/* Initial prd values -- origin, in_direction already set */
		prd.depth = 0;
		prd.done = false;
		prd.throughput = make_float3(1.0f);
		prd.pdf = 1.0f;
		prd.color = make_float3(0.0f);
		prd.Sample = CALLABLE_COUNT; /* I.e., invalid */

		uint32_t u0, u1;
		packPointer(&prd, u0, u1);

		/* We keep track of whether the previous hit was specular */
		bool previousHitSpecular = false;

		/* === Iterative path tracing loop === */
		while (true)
		{
			/* Trace the primary ray */
			optixTrace(
				optixLaunchParams.traversable,
				prd.origin,
				prd.in_direction,
				0.0f, /* tMin */
				1e20f, /* tMax */
				0.0f, /* ray time */
				OptixVisibilityMask(255),
				OPTIX_RAY_FLAG_DISABLE_ANYHIT,
				RAY_TYPE_RADIANCE, /* SBT offset */
				RAY_TYPE_COUNT, /* SBT stride */
				RAY_TYPE_RADIANCE, /* miss SBT index */
				u0, u1 /* packed pointer to our PRD */
			);
			prd.depth++;

			/* If the ray has terminated (e.g., hit background), end */
			if (prd.done)
			{
				/* Note: Throughput already has already been multiplied by background color */
				prd.color += prd.throughput;
				break;
			}


			/* If this is the first bounce and we hit a light or if we just had a specular hit, we add light emission */
			if ((prd.depth == 1 || previousHitSpecular) && prd.Sample == CALLABLE_DIFFUSE_LIGHT_SAMPLE)
			{
				prd.color += prd.throughput;
			}

			/* Now we can update with the current hit's specular bool */
			previousHitSpecular = prd.specular;

			/* Importance sample the lights */
			prd.color += prd.throughput * ImportanceSampleLight(prd);

			/* If max depth == 0, we use russian roulette to determine path termination */
			if (optixLaunchParams.maxDepth == 0)
			{
				/*
				 * We do not start russian roulette path termination until after first
				 * 3 bounces to make sure we can get at least some indirect lighting...
				 */
				if (prd.depth > 3)
				{
					/* Clamp russian roulette to 0.99f to prevent inf bounces for materials that do not absorb any light */
					float p = min(prd.pdf, 0.99f);
					if (prd.random() > p)
					{
						break;
					}
					prd.pdf /= p;
				}
			}
			/* Not using RR, terminate the random walk if we're at/past the max depth */
			else if (prd.depth >= optixLaunchParams.maxDepth) break;
		}
	}


	__forceinline__ __device__ void Integrate(PRD_Radiance& prd)
	{
		switch (optixLaunchParams.integrator)
		{
		case INTEGRATOR_TYPE_PATH:
		{
			PathIntegrator(prd);
			break;
		}
		}
	}


	/*
	 * The primary ray gen program that is called on Optix::Render()
	 */
	extern "C" __global__ void __raygen__renderFrame()
	{
		/* Get pixel position and framebuffer index */
		const int ix = optixGetLaunchIndex().x;
		const int iy = optixGetLaunchIndex().y;
		const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;

		/* Get the current frame's frameID -- i.e., which render call is this? */
		const int accumID = optixLaunchParams.frame.frameID;

		/* Initialize per-ray data */
		PRD_Radiance prd;

		/* Random seed is current frame count * frame size + current (1D) pixel position such that every pixel for every accumulated frame has a unique seed. */
		prd.random.Init(accumID * optixLaunchParams.frame.size.x * optixLaunchParams.frame.size.y + iy * optixLaunchParams.frame.size.x + ix, optixLaunchParams.sampler, optixLaunchParams.nStrata);

		const int numPixelSamples = optixLaunchParams.frame.samples; /* N Pixel samples for this render call */
		float3 pixelColor = make_float3(0.0f); /* Accumulated color for all pixel samples for this call */
		float3 pixelNormal = make_float3(0.0f); /* Accumulated normals for all pixel samples for this call */
		float3 pixelAlbedo = make_float3(0.0f); /* Accumulated albedo for all pixel samples for this call */
		for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
		{
			/* Determine the screen sampling position and generate corresponding camera ray */
			float2 screen = GenerateScreenPosition(ix, iy, prd.random);
			GenerateCameraRay(prd, screen);

			/* Run the integrator for this sample -- result is stored in prd.color */
			Integrate(prd);

			/* Set NaNs to 0 */
			if (prd.color.x != prd.color.x) prd.color.x = 0.0f;
			if (prd.color.y != prd.color.y) prd.color.y = 0.0f;
			if (prd.color.z != prd.color.z) prd.color.z = 0.0f;

			pixelColor += prd.color;
			pixelNormal += prd.normal;
			pixelAlbedo += prd.albedo;
		}

		/* Determine average color for this call. Cap to prevent speckles (even though this breaks pbr condition) */
		const float cap = 1e2f;
		const float cr = min(pixelColor.x / numPixelSamples, cap);
		const float cg = min(pixelColor.y / numPixelSamples, cap);
		const float cb = min(pixelColor.z / numPixelSamples, cap);
		const float4 ccolor = make_float4(cr, cg, cb, 1.0f);

		/* Determine the average albedo and normal for this call */
		pixelAlbedo = pixelAlbedo / numPixelSamples;
		const float4 albedo = make_float4(pixelAlbedo.x, pixelAlbedo.y, pixelAlbedo.z, 1.0f);

		pixelNormal = pixelNormal / numPixelSamples;
		const float4 normal = make_float4(pixelNormal.x, pixelNormal.y, pixelNormal.z, 1.0f);

		/* Get the current pixel's previously accumulated color, albedo, normal */
		float4 acolor = make_float4(0.0f);
		float4 aalbedo = make_float4(0.0f);
		float4 anormal = make_float4(0.0f);
		if (accumID > 0)
		{
			acolor = optixLaunchParams.frame.colorBuffer[fbIndex];
			aalbedo = optixLaunchParams.frame.albedoBuffer[fbIndex];
			anormal = optixLaunchParams.frame.normalBuffer[fbIndex];
		}

		/* Determine the new accumulated color, albedo, and normal */
		float4 tcolor = (ccolor + accumID * acolor) / (accumID + 1);
		tcolor = make_float4(min(tcolor.x, 1.0f), min(tcolor.y, 1.0f), min(tcolor.z, 1.0f), 1.0f);

		float4 talbedo = (albedo + accumID * aalbedo) / (accumID + 1);
		float4 tnormal = (normal + accumID * anormal) / (accumID + 1);

		/* Update the buffers */
		optixLaunchParams.frame.colorBuffer[fbIndex] = tcolor;
		optixLaunchParams.frame.albedoBuffer[fbIndex] = talbedo;
		optixLaunchParams.frame.normalBuffer[fbIndex] = tnormal;
	}

} /* namespace otx */